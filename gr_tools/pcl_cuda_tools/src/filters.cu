#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C"
{
  __device__
  int getGlobalIdx_1D_1D(){
    return blockIdx.x *blockDim.x + threadIdx.x;
  }

  __device__
  int getGlobalIdx_2D_2D(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
  }
    __global__
    void filter_passthrough_kernel(float *x, float *y, float *z, float min_limit, float max_limit){
      //__shared__ int s[256];
      //int idx = blockIdx.x * blockDim.x + threadIdx.x;
      //int index = blockIdx.x * blockDim.x + threadIdx.x;
      //int stride = blockDim.x * gridDim.x;
      int index = getGlobalIdx_2D_2D();

      if (index > 1024*1024)
      printf("%d \n", index  );

      if (max_limit > x[index] > min_limit){
        x[index] = -1;
      }
      if (max_limit > y[index] > min_limit){
        y[index] = -1;
      }
      if (max_limit > z[index] > min_limit){
        z[index] = -1;
      }
    }

    void free_memory(float *x){
      hipFree(x);
    }

    int apply_cuda_filter(float *o_x, float *o_y, float *o_z, float min_limit, float max_limit){
      // initialize x array on the host
      float * x, * y, *z;
      int size = 1024*1024;//sizeof(o_x)/sizeof(float);
      // Allocate Unified Memory – accessible from CPU or GPU
      hipMallocManaged(&x, size*sizeof(float));
      hipMallocManaged(&y, size*sizeof(float));
      hipMallocManaged(&z, size*sizeof(float));
      printf("A\n");
      hipMemcpy(x, o_x, size*sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(y, o_x, size*sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(z, o_x, size*sizeof(float), hipMemcpyHostToDevice);
      printf("b\n");

      int nthreads = 32;
      dim3 threads (nthreads,nthreads);
      int nblocks =  16;//ceil(size / nthreads)/8;//size/ nthreads -1;
      //memset(t, 0x00, nthreads);
      dim3 blocks(nblocks,nblocks);
      // First param blocks
      // Second param number of threads
      //  blocks, threads each
      printf("C %d\n", nblocks);

      filter_passthrough_kernel<<<blocks,threads>>>(x,y,z, min_limit, max_limit);
      hipDeviceSynchronize(); // to print results
      //cudaMemcpy(tr, t, sizeof(x), cudaMemcpyDeviceToHost);
      hipMemcpy(o_x, x, size*sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(o_y, y, size*sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(o_z, z, size*sizeof(float), hipMemcpyDeviceToHost);

      free_memory(x);
      free_memory(y);
      free_memory(z);
      printf("size %d\n",size );

      return 1;
    }
}
