#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C"
{
  __device__
  int getGlobalIdx_1D_1D(){
    return blockIdx.x *blockDim.x + threadIdx.x;
  }

  __device__
  int getGlobalIdx_2D_2D(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
  }
    __global__
    void filter_passthrough_kernel(float *z, bool *b, float min_limit, float max_limit, float filter_value, int size){
      //__shared__ int s[256];
      //int idx = blockIdx.x * blockDim.x + threadIdx.x;
      //int index = blockIdx.x * blockDim.x + threadIdx.x;
      //int stride = blockDim.x * gridDim.x;
      int index = getGlobalIdx_1D_1D();
      if (index >= size){
        //printf("%d\n", index );
        return;
      }


      /*
      if (max_limit < x[index] || x[index] < min_limit){
        //x[index] = filter_value;
        //y[index] = filter_value;
        b[index] = true;
      }
      if (max_limit < y[index] || y[index] < min_limit){
        //x[index] = filter_value;
        //y[index] = filter_value;
        b[index] = true;
      }
      */
      if (max_limit < z[index] || z[index] < min_limit){
        //x[index] = filter_value;
        //y[index] = filter_value;
        b[index] = true;
      }

      /*
      if (isnan(z[index])){
        b[index] = true;
      }

      if (isnan(x[index])){
        b[index] = true;
      }

      if (isnan(y[index])){
        b[index] = true;
      }
      */
    }

    int apply_cuda_filter(float *o_z, bool *o_b, float min_limit, float max_limit, float filter_value, int size){
      // initialize x array on the host
      float *z;
      //float *y;
      //float *x;
      bool *b;
      // Allocate Unified Memory – accessible from CPU or GPU
      //cudaMallocManaged(&x, size*sizeof(float));
      //cudaMallocManaged(&y, size*sizeof(float));
      hipMallocManaged(&z, size*sizeof(float));
      //cudaMallocManaged(&y, size*sizeof(float));
      //cudaMallocManaged(&x, size*sizeof(float));
      hipMallocManaged(&b, size*sizeof(bool));

      printf("min limit %f", min_limit);
      printf("max limit %f", max_limit);
      //cudaMemcpy(x, o_x, size*sizeof(float), cudaMemcpyHostToDevice);
      //cudaMemcpy(y, o_y, size*sizeof(float), cudaMemcpyHostToDevice);
      hipMemcpy(z, o_z, size*sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(b, o_b, size*sizeof(bool), hipMemcpyHostToDevice);


      int ngrid;      // The launch configurator returned block size
      int nblocks;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch
      //cudaOccupancyMaxPotentialBlockSize(&nblocks, &nthreads, filter_passthrough_kernel, 0, 0);
      ngrid = 1024;
      dim3 grid (ngrid);
      nblocks = ceil((size+ngrid -1)/ngrid);

      //nblocks = (size + nblocks -1)/nblocks;
      printf("\n A %d %d %d\n", size, nblocks, ngrid);

      //int nthreads = 1024;
      //int nblocks = ceil(size/nthreads);//7ceil(size / nthreads);//size/ nthreads -1;
      //memset(t, 0x00, nthreads);
      dim3 blocks(nblocks);
      // First param blocks
      // Second param number of threads
      //  blocks, threads each
      printf("C %d %d\n", nblocks, ngrid);
      printf("Deeee %d\n", size);

      //printf("OK %d", max_thread - size);
      //filter_passthrough_kernel<<<blocks,threads>>>(z,b,min_limit, max_limit, filter_value, size);
      filter_passthrough_kernel<<<blocks,grid>>>(z,b,min_limit, max_limit, filter_value, size);
      hipDeviceSynchronize(); // to print results
      //cudaMemcpy(o_x, x, size*sizeof(float), cudaMemcpyDeviceToHost);
      //cudaMemcpy(o_y, y, size*sizeof(float), cudaMemcpyDeviceToHost);
      //cudaMemcpy(o_z, z, size*sizeof(float), cudaMemcpyDeviceToHost);
      hipMemcpy(o_b, b, size*sizeof(bool), hipMemcpyDeviceToHost);

      //free_memory(x);
      //free_memory(y);
      hipFree(z);
      hipFree(b);
      printf("DONE!!!!");


      return 1;
    }
}
